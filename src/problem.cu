#include "hip/hip_runtime.h"
#include <iostream>
#include <cassert>
#include <kernels.h>
#include <hipsolver.h>
#include <errs.h>
#include <random>
#include <cstdio>
#include <iomanip>
#include <string>
using namespace std;

/* extern "C" {
    void dgemm_(char*, char*, int*, int*, int*, double*, double*, int*, double*, int*, double*, double*, int*);
} */

Problem::Problem() {
}

Problem::Problem(Data &data, Parameter &parameter, Setting &setting) {
}

void Problem::randinit(std::uint64_t n, std::uint64_t m) {

    std::random_device r;
    std::mt19937 gen(r());
    std::uniform_real_distribution<> distr(1.0, 2.0);

    for(int i = 0; i < data.n * data.m; i++)
        this->data.A.values[i] = distr(gen);
};

void Problem::print() const {
    cout << "The solution vector x is:" << endl;
    for (int i = 0; i < data.n; i++) {
        cout << solution.x[i] << " ";
    }
    cout << endl;
    cout << endl;

    cout << "The solution vector y is:" << endl;
    for (int i=0; i < data.m; i++) {
        cout << solution.y[i] << " ";
    }
};

void Problem::osqp() {
};

void Problem::cuda_allocate(Problem *problem_new) {
    cuErrChk(hipMalloc((void **)&(problem_new->data.q), sizeof(double)*data.n));
    cuErrChk(hipMalloc((void **)&(problem_new->data.P.values), sizeof(double)*data.P.nrows*data.P.ncols));
    cuErrChk(hipMalloc((void **)&(problem_new->data.A.values), sizeof(double)*data.A.nrows*data.A.ncols));
    cuErrChk(hipMalloc((void **)&(problem_new->data.l), sizeof(double)*data.m));
    cuErrChk(hipMalloc((void **)&(problem_new->data.u), sizeof(double)*data.m));

    cuErrChk(hipMalloc((void **)&(problem_new->solution.x), sizeof(double)*data.n));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.z), sizeof(double)*data.m));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.y), sizeof(double)*data.m));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.primal), sizeof(double)));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.dual), sizeof(double)));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.x_v), sizeof(double)*(data.m+data.n)));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.z_tilde), sizeof(double)*data.m));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.deltax), sizeof(double)*data.n));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.deltay), sizeof(double)*data.m));

    cuErrChk(hipMalloc((void **)&(problem_new->solution.norm1_vec), sizeof(double)*data.m));
    cuErrChk(hipMalloc((void **)&(problem_new->solution.norm2_vec), sizeof(double)*data.n));

    cuErrChk(hipMalloc((void **) &(problem_new->data.sol_con.values), sizeof(double)*(data.m+data.n)*(data.m+data.n)));
    cuErrChk(hipMalloc((void **) &(problem_new->data.vec_con), sizeof(double)*(data.m+data.n)));

    cuErrChk(hipMemcpy(problem_new->data.q, data.q, sizeof(double)*data.n, hipMemcpyHostToDevice));
    cuErrChk(hipMemcpy(problem_new->data.l, data.l, sizeof(double)*data.m, hipMemcpyHostToDevice));
    cuErrChk(hipMemcpy(problem_new->data.u, data.u, sizeof(double)*data.m, hipMemcpyHostToDevice));

    cuErrChk(hipMemcpy(problem_new->data.P.values, data.P.values, sizeof(double)*data.P.nrows*data.P.ncols, hipMemcpyHostToDevice));
    cuErrChk(hipMemcpy(problem_new->data.A.values, data.A.values, sizeof(double)*data.A.nrows*data.A.ncols, hipMemcpyHostToDevice));

    cuErrChk(hipMemcpy(problem_new->solution.x, solution.x, sizeof(double)*data.n, hipMemcpyHostToDevice));
    cuErrChk(hipMemcpy(problem_new->solution.y, solution.y, sizeof(double)*data.m, hipMemcpyHostToDevice));
    cuErrChk(hipMemcpy(problem_new->solution.z, solution.z, sizeof(double)*data.m, hipMemcpyHostToDevice));
    cuErrChk(hipMemcpy(problem_new->solution.x_v, solution.x_v, sizeof(double)*(data.n+data.m), hipMemcpyHostToDevice));

    cuErrChk(hipMemcpy(problem_new->solution.norm1_vec, solution.z, sizeof(double)*data.m, hipMemcpyHostToDevice));
    cuErrChk(hipMemcpy(problem_new->solution.norm2_vec, data.q, sizeof(double)*data.n, hipMemcpyHostToDevice));
};

void Problem::checknorm(hipblasHandle_t cublas_handle, Problem *problem_new) {
    double scalar = 1.0;

    dim3 grid_size, block_size;
    size_t shmem_size;
    grid_size.x = (int)((data.m + 127) / 128);
    grid_size.y = (int)((data.n + 127) / 128);
    block_size.x = 16;
    block_size.y = 16;
    shmem_size = 1;

    cuErrChk(hipblasDgemv(cublas_handle, HIPBLAS_OP_N, data.m, data.n, &scalar, problem_new->data.A.values, data.m, problem_new->solution.x, 1, &scalar, problem_new->solution.norm1_vec, 1));
    vecminus <<< grid_size, block_size, shmem_size >>> (problem_new->solution.norm1_vec, problem_new->solution.z, data.m);
    cuErrChk(hipblasDnrm2(cublas_handle, data.m, problem_new->solution.norm1_vec, 1, solution.norm1));
    // norm1 = norm(A*x-z)

    cuErrChk(hipblasDgemv(cublas_handle, HIPBLAS_OP_T, data.n, data.m, &scalar, problem_new->data.A.values, data.n, problem_new->solution.y, 1, &scalar, problem_new->solution.norm2_vec, 1));
    cuErrChk(hipblasDgemv(cublas_handle, HIPBLAS_OP_N, data.n, data.n, &scalar, problem_new->data.P.values, data.n, problem_new->solution.x, 1, &scalar, problem_new->solution.norm2_vec, 1));
    cuErrChk(hipblasDnrm2(cublas_handle, data.n, problem_new->solution.norm2_vec, 1, solution.norm2));
    // norm2 = norm(P*x+q+A*y)
};

void Problem::cuda_free(Problem *problem_new) {
    cuErrChk(hipFree(problem_new->data.q));
    cuErrChk(hipFree(problem_new->data.P.values));
    problem_new->data.P.values = nullptr;
    cuErrChk(hipFree(problem_new->data.A.values));
    problem_new->data.A.values = nullptr;
    cuErrChk(hipFree(problem_new->data.l));
    cuErrChk(hipFree(problem_new->data.u));

    cuErrChk(hipFree(problem_new->solution.x));
    cuErrChk(hipFree(problem_new->solution.z));
    cuErrChk(hipFree(problem_new->solution.y));
    cuErrChk(hipFree(problem_new->solution.x_v));
    cuErrChk(hipFree(problem_new->solution.z_tilde));
    cuErrChk(hipFree(problem_new->solution.deltax));
    cuErrChk(hipFree(problem_new->solution.deltay));
    cuErrChk(hipFree(problem_new->solution.primal));
    cuErrChk(hipFree(problem_new->solution.dual));    

    cuErrChk(hipFree(problem_new->solution.norm1_vec));
    cuErrChk(hipFree(problem_new->solution.norm2_vec));
    cuErrChk(hipFree(problem_new->data.sol_con.values));
    problem_new->data.sol_con.values = nullptr;
    cuErrChk(hipFree(problem_new->data.vec_con));
};

/* Function to implement CUDA version OSQP
 * Minimize 1/2 * x^T * P * x + q^T * x
 * Subject to l <= A^T * x <= u
 */
void Problem::cu_all_osqp() {
    Problem *problem_new = new Problem();
    cuda_allocate(problem_new);

    hipsolverHandle_t solver_handle;
    cuErrChk(hipsolverDnCreate(&solver_handle));

    hipblasHandle_t cublas_handle;
    cuErrChk(hipblasCreate(&cublas_handle));

    hipStream_t stream = nullptr;
    cuErrChk(hipStreamCreate(&stream));
    cuErrChk(hipsolverSetStream(solver_handle, stream));

    dim3 grid_size, block_size;
    size_t shmem_size;
    grid_size.x = (int)((data.m + 127) / 128);
    grid_size.y = (int)((data.n + 127) / 128);
    block_size.x = 16;
    block_size.y = 16;
    shmem_size = 1;

    int size = data.m + data.n;
    hipDeviceSynchronize();

    concatenateMatricesKernel<<<grid_size, block_size, shmem_size>>>(problem_new->data.sol_con.values, problem_new->data.P.values,
                problem_new->data.A.values, data.m, data.n, parameter.rho, parameter.sigma);

    // cuErrChk(hipMemcpy(data.sol_con.values, problem_new->data.sol_con.values, sizeof(double)*(data.m+data.n)*(data.m+data.n), hipMemcpyDeviceToHost));
    int Lwork = 0;
    cuErrChk(hipsolverDnDgetrf_bufferSize(solver_handle, size, size, problem_new->data.sol_con.values, size, &Lwork));
    double* d_Work;
    cuErrChk(hipMalloc((void**)&d_Work, sizeof(double) * Lwork));
    int* d_info;
    cuErrChk(hipMalloc((void**)&d_info, sizeof(int)));
    int* devIpiv;
    cuErrChk(hipMalloc((void**)&devIpiv, sizeof(int) * size));
    // LU factorization
    cuErrChk(hipsolverDnDgetrf(solver_handle, size, size, problem_new->data.sol_con.values, size, d_Work, devIpiv, d_info));
    
    hipDeviceSynchronize();

    int iters = 1;

    while (iters < setting.max_iter && (*solution.norm1 >= setting.eps_prim || *solution.norm2 >= setting.eps_dual)) {

        process_xv <<< grid_size, block_size, shmem_size >>> (parameter.sigma, parameter.rho, problem_new->solution.x, problem_new->solution.y, problem_new->solution.z, problem_new->data.q, data.n, data.m, problem_new->solution.x_v);
        // Solve the linear system [[P+rho*I, A^T], [A, -(1/rho)*I]]*x_v = [rho*x-q, z-(1/rho)*y];
        cuErrChk(hipDeviceSynchronize());
        cuErrChk(hipsolverDnDgetrs(solver_handle, HIPBLAS_OP_N, size, 1, problem_new->data.sol_con.values, size, devIpiv, problem_new->solution.x_v, size, d_info));
        cuErrChk(hipDeviceSynchronize());

        // z_tilde = z + (1/rho) * (vecAdd <<< grid_size, block_size, shmem_size  >>> (v, -y));
        // x_new = alpha*x+(1-alpha) * x;
        // z_new = multiply(alpha*z_tilde + (1-alpha)*z + (1/rho)*y);
        // y_new = y + rho * (alpha * z_tilde + (1-alpha) * z - z*new);

        update <<< grid_size, block_size, shmem_size >>> (problem_new->solution.x,
            problem_new->solution.y, problem_new->solution.z, problem_new->solution.x_v, 
            problem_new->solution.z_tilde, problem_new->solution.primal, problem_new->solution.dual,
            problem_new->solution.deltax, problem_new->solution.deltay, parameter.rho, 
                parameter.alpha, problem_new->data.l, problem_new->data.u, data.m, data.n);
        /*cuErrChk(hipMemcpy(solution.primal, problem_new.solution.primal, sizeof(double), hipMemcpyDeviceToHost));
        cuErrChk(hipMemcpy(solution.dual, problem_new.solution.dual, sizeof(double), hipMemcpyDeviceToHost));

        if (solution.primal == 0.0) {
            continue;
        } else {
            check_primal_infeasibility <<< grid_size, block_size, shmem_size >>> (&(problem_new.solution));
        }

        if (solution.dual == 0.0) {
            continue;
        } else {
            check_dual_infeasibility <<< grid_size, block_size, shmem_size >>> (&(problem_new.solution, problem_new.data.u, problem_new.data.l, data.m));
        }

        cuErrChk(hipMemcpy(solution.primal, problem_new.solution.primal, sizeof(double), hipMemcpyDeviceToHost));
        cuErrChk(hipMemcpy(solution.dual, problem_new.solution.dual, sizeof(double), hipMemcpyDeviceToHost));

        if (solution.primal != 0 || solution.dual != 0) {
            cout << "Infeasibility detected!" << endl;
            break;
        }*/
        
        checknorm(cublas_handle, problem_new);
        iters++;
    }

    cuErrChk(hipMemcpy(solution.x, problem_new->solution.x, sizeof(double)*data.n, hipMemcpyDeviceToHost));
    cuErrChk(hipMemcpy(solution.y, problem_new->solution.y, sizeof(double)*data.m, hipMemcpyDeviceToHost));

    cuda_free(problem_new);

    cuErrChk(hipFree(d_Work));
    cuErrChk(hipFree(d_info));
    cuErrChk(hipFree(devIpiv));

    cuErrChk(hipStreamDestroy(stream));
    cuErrChk(hipsolverDnDestroy(solver_handle));
    cuErrChk(hipblasDestroy(cublas_handle));

    delete problem_new;
};

void Problem::cu_osqp() {
};
