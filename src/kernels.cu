#include "hip/hip_runtime.h"
#include <kernels.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>

/*
   * Global device kernel function to implement semi-ring
   * DGEMM.
   */

__global__ void vecAdd(double *a, double *b, double *c, int n)
{
    // Get our global thread ID
    int id = blockIdx.x*blockDim.x+threadIdx.x;
 
    // Make sure we do not go out of bounds
    if (id < n)
        c[id] = a[id] + b[id];
}

__global__ 
void osqp_kernel(uint64_t const *__restrict__ nr_A,
        uint64_t const *__restrict__ nc_A,
        double const *__restrict__ A,
        uint64_t const *__restrict__ nc_B,
        double const *__restrict__ B,
        double *__restrict__ C) {
}

__global__ 
void concatenateMatricesKernel(Problem* prob,
                        int m, int n, double rho) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Concatenate P + rho*I
    if (idx < n * n) {
        int i = idx / n;
        int j = idx % n;
        prob->data.sol_con.values[idx] = prob->data.P.values[idx] + ((i == j) ? rho : 0.0);
    }

    // Concatenate A^T
    if (idx >= n * n && idx < n * n + n * m) {
        int i = (idx - n * n) / m;
        int j = (idx - n * n) % m;
        prob->data.sol_con.values[idx] = prob->data.A.values[j * n + i];
    }

    // Concatenate A
    if (idx >= n * n + n * m && idx < n * n + 2 * n * m) {
        int i = (idx - n * n - n * m) / n;
        int j = (idx - n * n - n * m) % n;
        prob->data.sol_con.values[idx] = prob->data.A.values[i * n + j];
    }

    // Concatenate -(1/rho)*I
    if (idx >= n * n + 2 * n * m && idx < n * n + 3 * n * m) {
        int i = (idx - n * n - 2 * n * m) / m;
        int j = (idx - n * n - 2 * n * m) % m;
        prob->data.sol_con.values[idx] = ((i == j) ? -1.0 / rho : 0.0);
    }
}

__global__ void concatenateVectorsKernel(Problem* prob, double sigma, double rho, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size) {
        // Concatenate [sigma*x - q]
        prob->data.vec_con[idx] = sigma * prob->solution.x[idx] - prob->data.q[idx];

        // Concatenate [z - (1/rho)*y]
        prob->data.vec_con[idx + size] = prob->solution.z[idx] - prob->solution.y[idx] / rho;
    }
}

__global__ void update(double *x, double *y, double *z, double *x_v, double *z_tilde, double *primal, double* dual,
        double *deltax, double *deltay, double rho, double alpha, double *l, double *u, int m, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Compute z_tilde = z_k + (1/rho) * (v - y)
    // x_new = alpha*x+(1-alpha) * x;
    // z_new = multiply(alpha*z_tilde + (1-alpha)*z + (1/rho)*y);
    // y_new = y + rho * (alpha * z_tilde + (1-alpha) * z - z*new);
    __shared__ double *delta_x_norm;
    __shared__ double *delta_y_norm;

    // Initialize shared variables to 0
    if (threadIdx.x == 0) {
        *delta_x_norm = 0.0;
        *delta_y_norm = 0.0;
    }

    __syncthreads();

    if (idx < n) {
        z_tilde[idx] = z[idx] + (x_v[idx + m] - y[idx]) / rho;
        z_tilde[idx] = alpha * z_tilde[idx] + (1-alpha) * z[idx];

        if (z_tilde[idx]+(1/rho)*y[idx] < l[idx]) {
                z[idx] = l[idx];
        } else if (z_tilde[idx] + (1/rho)*y[idx] > u[idx]) {
                z[idx] = u[idx];
        } else {
                z[idx] = z_tilde[idx];
        }

        deltay[idx] = (1/rho) * (z_tilde[idx] - z[idx]);
        y[idx] = y[idx] + deltay[idx];

        // Atomic add to accumulate the result
        *delta_y_norm += deltay[idx];
    }

    if (n <= idx < n+m) {
        deltax[idx-n] = alpha * (x_v[idx-n] - x[idx-n]);
        x[idx-n] = x[idx-n] + deltax[idx-n];

        // Atomic add to accumulate the result
        *delta_x_norm += deltax[idx-n];

    }

    __syncthreads();

    // Store the final result in global memory
    if (threadIdx.x == 0) {
        // Atomic add to accumulate the final result
        *primal += *delta_x_norm;
        *dual += *delta_y_norm;
    }
    __syncthreads();
}

__global__ 
void check_primal_infeasibility() {

}

__global__ 
void check_dual_infeasibility(double *x, double *y, double *dual, double *u, double *l, int m) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ double vec_sum;
    __syncthreads();

    if (idx < m) {
        if (y[idx] < 0) {
            vec_sum += y[idx] * l[idx];
        } else if (y[idx] > 0) {
            vec_sum += y[idx] * u[idx];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        *dual += vec_sum;
    }

    __syncthreads();
}