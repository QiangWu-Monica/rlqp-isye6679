#include "hip/hip_runtime.h"
#include <kernels.h>
#include <vector>
#include <hip/hip_runtime.h>
#include <float.h>
#include <stdio.h>

/*
   * Global device kernel function to implement semi-ring
   * DGEMM.
   */

__global__ void vecminus(double* vec, double* minvec, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;    

    if (idx < size && idy == 1) {
        vec[idx] = vec[idx] - minvec[idx];
    }

    __syncthreads();
}

__global__ void process_xv(double sigma, double rho, double *x, double *y, double *z, double *q, int n, int m, double *x_v) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;

    if (idx < n && idy == 1) {
        x_v[idx] = sigma * x[idx] - q[idx];
    } else if (n <= idx && idx < n+m) {
        x_v[idx] = z[idx-n] - (1/rho) * y[idx-n];
    }

    __syncthreads();
}

__global__ 
void osqp_kernel(uint64_t const *__restrict__ nr_A,
        uint64_t const *__restrict__ nc_A,
        double const *__restrict__ A,
        uint64_t const *__restrict__ nc_B,
        double const *__restrict__ B,
        double *__restrict__ C) {
}

__global__ 
void concatenateMatricesKernel(double* sol_con, double* P, double* A,
                        int m, int n, double rho, double sigma) {
    int idx = threadIdx.x + blockDim.x * blockIdx.x;
    int idy = threadIdx.y + blockDim.y * blockIdx.y;

    if (idx < m+n && idy < m+n) {
        // Concatenate P + rho*I
        if (idx < n && idy < n) {
            sol_con[idy*(n+m)+idx] = P[idy*n+idx] + ((idx == idy) ? sigma : 0.0);
        } else if (n <= idy && idy < m+n && idx < n) { // Concatenate A^T
            sol_con[idy*(n+m)+idx] = A[idx*m+(idy-n)];
        } else if (n <= idx && idx < n+m && idy < n) { // Concatenate A
            sol_con[idy*(n+m)+idx] = A[idy*m+(idx-n)];
        } else if (n <= idx && n <= idy && idx < n+m && idy < n+m) { // Concatenate -(1/rho)*I
            sol_con[idy*(n+m)+idx] = ((idx == idy) ? -1.0/rho : 0.0);
        }
    }

    __syncthreads();
}

__global__ void concatenateVectorsKernel(Problem* prob, double sigma, double rho, int size) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (idx < size) {
        // Concatenate [sigma*x - q]
        prob->data.vec_con[idx] = sigma * prob->solution.x[idx] - prob->data.q[idx];

        // Concatenate [z - (1/rho)*y]
        prob->data.vec_con[idx + size] = prob->solution.z[idx] - prob->solution.y[idx] / rho;
    }

    __syncthreads();
}

__global__ void update(double *x, double *y, double *z, double *x_v, double *z_tilde, double *primal, double* dual,
        double *deltax, double *deltay, double rho, double alpha, double *l, double *u, int m, int n) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // Compute z_tilde = z_k + (1/rho) * (v - y)
    // x_new = alpha*x+(1-alpha) * x;
    // z_new = multiply(alpha*z_tilde + (1-alpha)*z + (1/rho)*y);
    // y_new = y + rho * (alpha * z_tilde + (1-alpha) * z - z*new);
    // __shared__ double *delta_x_norm;
    // __shared__ double *delta_y_norm;

    // Initialize shared variables to 0
    // if (threadIdx.x == 0) {
    //     *delta_x_norm = 0.0;
    //     *delta_y_norm = 0.0;
    // }

    __syncthreads();

    if (idx < m) {
        z_tilde[idx] = z[idx] + (x_v[idx + n] - y[idx]) / rho;
        z_tilde[idx] = alpha * z_tilde[idx] + (1-alpha) * z[idx];

        if (z_tilde[idx]+(1/rho)*y[idx] < l[idx]) {
                z[idx] = l[idx];
        } else if (z_tilde[idx] + (1/rho)*y[idx] > u[idx]) {
                z[idx] = u[idx];
        } else {
                z[idx] = z_tilde[idx];
        }

        deltay[idx] = (1/rho) * (z_tilde[idx] - z[idx]);
        y[idx] = y[idx] + deltay[idx];

        // Atomic add to accumulate the result
        // *delta_y_norm += deltay[idx];
    } else if (m <= idx && idx < m+n) {
        deltax[idx-m] = alpha * (x_v[idx-m] - x[idx-m]);
        x[idx-m] = x[idx-m] + deltax[idx-m];

        // Atomic add to accumulate the result
        // *delta_x_norm += deltax[idx-n];

    }

    __syncthreads();

    // Store the final result in global memory
    // if (threadIdx.x == 0) {
        // Atomic add to accumulate the final result
    //     *primal += *delta_x_norm;
    //     *dual += *delta_y_norm;
    // }
    // __syncthreads();
}

__global__ 
void check_primal_infeasibility() {

}

__global__ 
void check_dual_infeasibility(double *x, double *y, double *dual, double *u, double *l, int m) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    __shared__ double vec_sum;
    __syncthreads();

    if (idx < m) {
        if (y[idx] < 0) {
            vec_sum += y[idx] * l[idx];
        } else if (y[idx] > 0) {
            vec_sum += y[idx] * u[idx];
        }
    }

    __syncthreads();

    if (threadIdx.x == 0) {
        *dual += vec_sum;
    }

    __syncthreads();
}